#include "hip/hip_runtime.h"
#include "alloc3d_gpu.h"
#include "transfer3d_gpu.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void gpu_par_kernel(int N, double ***u1, double ***u2) {
    int i = 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int j = 1 + blockIdx.y * blockDim.y + threadIdx.y;
    int k = 1 + blockIdx.z * blockDim.z + threadIdx.z;

    if (i <= N && j <= N && k <= N) {
        double delta = 2.0 / (double)(N + 2);
        double delta2 = delta * delta;

        double x = -1.0 + (i * delta);
        double y = -1.0 + (j * delta);
        double z = -1.0 + (k * delta);
        double sum =
            u1[i - 1][j][k] +
            u1[i][j - 1][k] +
            u1[i][j][k - 1] +
            u1[i + 1][j][k] +
            u1[i][j + 1][k] +
            u1[i][j][k + 1];

        int mask = (x <= -0.375 && y <= -0.5 && -(2.0 / 3.0) <= z);
        double f = mask * 200.0;
        u2[i][j][k] = (sum + delta2 * f) / 6.0;
    }
}

int gpu_par(int N, int iter_max, double ***u_h) {
    int iter;

    double ***u1_d, ***u2_d;

    if ( (u1_d = d_malloc_3d_gpu(N+2, N+2, N+2)) == NULL ) {
        perror("array u1_d: allocation on gpu failed");
        exit(-1);
    }
    if ( (u2_d = d_malloc_3d_gpu(N+2, N+2, N+2)) == NULL ) {
        perror("array u2_d: allocation on gpu failed");
        exit(-1);
    }

    transfer_3d(u1_d, u_h, N+2, N+2, N+2, hipMemcpyHostToDevice);
    transfer_3d(u2_d, u_h, N+2, N+2, N+2, hipMemcpyHostToDevice);

    int blocks = ((N+1) + 7) / 8;
    dim3 dimGrid(blocks,blocks,blocks);
    dim3 dimBlock(8,8,8);

    for (iter = 0; iter < iter_max; ++iter) {
        gpu_par_kernel<<<dimGrid,dimBlock>>>(N, u1_d, u2_d);
        hipDeviceSynchronize();

        double ***tmp = u1_d;
        u1_d = u2_d;
        u2_d = tmp;
    }

    transfer_3d(u_h, u1_d, N+2, N+2, N+2, hipMemcpyDeviceToHost);

    free_gpu(u1_d);
    free_gpu(u2_d);

    return iter;
}
