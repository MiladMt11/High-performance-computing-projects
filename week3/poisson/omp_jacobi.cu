#include "hip/hip_runtime.h"
#include <math.h>
#include "alloc3d.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include "omp.h"

int cpu_jacobi_nonorm(
    int N,
    int iter_max,
    double ***u
) {
    int iter;
    double ***u1, ***u2;
    if ((u2 = d_malloc_3d(N + 2, N + 2, N + 2)) == NULL)
    {
        perror("array u2: allocation failed");
        exit(-1);
    }
    if ((u1 = d_malloc_3d(N + 2, N + 2, N + 2)) == NULL)
    {
        perror("array u1: allocation failed");
        exit(-1);
    }

    double delta = 2.0 / (double)(N + 2);
    double delta2 = delta * delta;

    // Copy over initial conditions.
    #pragma omp parallel for
    for (int i = 0; i < N + 2; ++i)
    {
        for (int j = 0; j < N + 2; ++j)
        {
            for (int k = 0; k < N + 2; ++k)
            {
                u1[i][j][k] = u[i][j][k];
                u2[i][j][k] = u[i][j][k];
            }
        }
    }


    for (iter = 0; iter < iter_max; ++iter)
    {
        #pragma omp parallel for
        for (int i = 1; i < N + 1; ++i)
        {
            double x = -1.0 + (i * delta);
            for (int j = 1; j < N + 1; ++j)
            {
                double y = -1.0 + (j * delta);
                for (int k = 1; k < N + 1; ++k)
                {
                    double z = -1.0 + (k * delta);
                    double sum =
                        u1[i - 1][j][k] +
                        u1[i][j - 1][k] +
                        u1[i][j][k - 1] +
                        u1[i + 1][j][k] +
                        u1[i][j + 1][k] +
                        u1[i][j][k + 1];

                    double f = (x <= -0.375 && y <= -0.5 && -(2.0 / 3.0) <= z) ? 200.0 : 0.0;
                    u2[i][j][k] = (sum + delta2 * f) / 6.0;
                }
            }
        }

        double ***utmp = u2;
        u2 = u1;
        u1 = utmp;
    }

    // Copy back results.
    for (int i = 0; i < N + 2; ++i)
    {
        for (int j = 0; j < N + 2; ++j)
        {
            for (int k = 0; k < N + 2; ++k)
            {
                u[i][j][k] = u2[i][j][k];
            }
        }
    }
    free(u1);
    free(u2);

    return iter;
}

int cpu_jacobi_norm(
    int N,
    int iter_max,
    double tolerance,
    double ***u
) {
    int iter;
    double ***u1, ***u2;
    if ((u2 = d_malloc_3d(N + 2, N + 2, N + 2)) == NULL)
    {
        perror("array u2: allocation failed");
        exit(-1);
    }
    if ((u1 = d_malloc_3d(N + 2, N + 2, N + 2)) == NULL)
    {
        perror("array u1: allocation failed");
        exit(-1);
    }

    double delta = 2.0 / (double)(N + 2);
    double delta2 = delta * delta;

    // Copy over initial conditions.
    #pragma omp parallel for
    for (int i = 0; i < N + 2; ++i)
    {
        for (int j = 0; j < N + 2; ++j)
        {
            for (int k = 0; k < N + 2; ++k)
            {
                u1[i][j][k] = u[i][j][k];
                u2[i][j][k] = u[i][j][k];
            }
        }
    }


    for (iter = 0; iter < iter_max; ++iter)
    {
        double norm = 0.0;
        #pragma omp parallel for reduction(+: norm)
        for (int i = 1; i < N + 1; ++i)
        {
            double x = -1.0 + (i * delta);
            for (int j = 1; j < N + 1; ++j)
            {
                double y = -1.0 + (j * delta);
                for (int k = 1; k < N + 1; ++k)
                {
                    double z = -1.0 + (k * delta);
                    double sum =
                        u1[i - 1][j][k] +
                        u1[i][j - 1][k] +
                        u1[i][j][k - 1] +
                        u1[i + 1][j][k] +
                        u1[i][j + 1][k] +
                        u1[i][j][k + 1];

                    double f = (x <= -0.375 && y <= -0.5 && -(2.0 / 3.0) <= z) ? 200.0 : 0.0;
                    u2[i][j][k] = (sum + delta2 * f) / 6.0;

                    double diff = u1[i][j][k] - u2[i][j][k];
                    norm += diff * diff;
                }
            }
        }

        if (norm < tolerance * tolerance)
            break;

        double ***utmp = u2;
        u2 = u1;
        u1 = utmp;
    }

    // Copy back results.
    for (int i = 0; i < N + 2; ++i)
    {
        for (int j = 0; j < N + 2; ++j)
        {
            for (int k = 0; k < N + 2; ++k)
            {
                u[i][j][k] = u2[i][j][k];
            }
        }
    }
    free(u1);
    free(u2);

    return iter;
}
