#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cblas.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hip/hip_runtime_api.h>

#define min(X, Y) ((X) < (Y) ? (X) : (Y))

extern "C"
{; // just for Ernie's IDE indentation config, please don't remove it :)

void matmult_nat(int m, int n, int k, double* A, double* B, double* C)
{
    // A(m,k) m - # of rows; k - # of columns
    // B(k,n) k - # of rows; n - # of columns
    // C(m,n) m - # of rows; n - # of columns
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (i = 0; i < m; i++) // i - row #
    {
        for (l = 0; l < k; l++) // k - row #
        {
            for (j = 0; j < n; j++) // j - col #
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_lib(int m, int n, int k, double* A, double* B, double* C)
{
    cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
}

void matmult_mkn(int m, int n, int k, double* A, double* B, double* C)
{
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (i = 0; i < m; i++)
    {
        for (l = 0; l < k; l++)
        {
            for (j = 0; j < n; j++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_mnk(int m, int n, int k, double* A, double* B, double* C)
{
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < n; j++)
        {
            for (l = 0; l < k; l++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_kmn(int m, int n, int k, double* A, double* B, double* C)
{
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (l = 0; l < k; l++)
    {
        for (i = 0; i < m; i++)
        {
            for (j = 0; j < n; j++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_knm(int m, int n, int k, double* A, double* B, double* C)
{
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (l = 0; l < k; l++)
    {
        for (j = 0; j < n; j++)
        {
            for (i = 0; i < m; i++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_nmk(int m, int n, int k, double* A, double* B, double* C)
{
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (j = 0; j < n; j++)
    {
        for (i = 0; i < m; i++)
        {
            for (l = 0; l < k; l++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_nkm(int m, int n, int k, double* A, double* B, double* C)
{
    int i, j, l;
    memset(C, 0, sizeof(double) * m * n);
    for (j = 0; j < n; j++)
    {
        for (l = 0; l < k; l++)
        {
            for (i = 0; i < m; i++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_blk(int m, int n, int k, double* A, double* B, double* C, int bs)
{
    int i_block, j_block, l_block;
    memset(C, 0, sizeof(double) * m * n);
    for (i_block = 0; i_block < m; i_block += bs)
    {
        for (l_block = 0; l_block < k; l_block += bs)
        {
            for (j_block = 0; j_block < n; j_block += bs)
            {
                int i, j, l, i_max, j_max, l_max;
                i_max = min(m, i_block + bs);
                j_max = min(n, j_block + bs);
                l_max = min(k, l_block + bs);
                for (i = i_block; i < i_max; i++)
                {
                    for (l = l_block; l < l_max; l++)
                    {
                        for (j = j_block; j < j_max; j++)
                        {
                            C[i * n + j] += A[i * k + l] * B[l * n + j];
                        }
                    }
                }
            }
        }
    }
}

__global__ void gpu1_kernel(int m, int n, int k, double* A, double* B, double* C)
{
    // A(m,k) m - # of rows; k - # of columns
    // B(k,n) k - # of rows; n - # of columns
    // C(m,n) m - # of rows; n - # of columns
    int i, j, l;
    for (i = 0; i < m; i++)
    {
        for (l = 0; l < k; l++)
        {
            for (j = 0; j < n; j++)
            {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

// The matrix sizes of A and B are m×k and k×n, respectively, so that C has size m×n
void matmult_gpu1(int m, int n, int k, double* A_h, double* B_h, double* C_h)
{
    // Allocate A_d, B_d, C_d
    double* A_d, * B_d, * C_d;
    hipMalloc((void**)&A_d, m * k * sizeof(double));
    hipMalloc((void**)&B_d, k * n * sizeof(double));
    hipMalloc((void**)&C_d, m * n * sizeof(double));

    // Transfer data
    hipMemcpy(A_d, A_h, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, k * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(C_d, 0, m * n * sizeof(int));

    // Launch kernel and synchronize
    gpu1_kernel << <1, 1 >> > (m, n, k, A_d, B_d, C_d);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(A_h, A_d, m * k * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(B_h, B_d, k * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C_h, C_d, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Free A_d, B_d, C_d
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

#define BLOCK_SIZE 8

__global__ void gpu2_kernel(int m, int n, int k, double* A, double* B, double* C, int bsx, int bsy)
{
    // A(m,k) m - # of rows; k - # of columns
    // B(k,n) k - # of rows; n - # of columns
    // C(m,n) m - # of rows; n - # of columns

    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int j = blockIdx.y * blockDim.y + threadIdx.y;

    int threadRowID, threadColID;
    threadRowID = blockIdx.x * blockDim.x + threadIdx.x;
    threadColID = blockIdx.y * blockDim.y + threadIdx.y;

    // /* ------------------------------------
    //    Print the thread's 2 dim grid ID
    //    ------------------------------------ */
    // printf("Blk: (%d,%d) Thread: (%d,%d) -> Row/Col = (%d,%d)\n",
    //     blockIdx.x, blockIdx.y,
    //     threadIdx.x, threadIdx.y,
    //     threadRowID, threadColID);

    // if (i * j >= m * n)
    //     return;
    if (threadColID >= n || threadRowID >= m)
        return;

    // if (i * j > 3000)
    // {
    // }

    for (int l = 0; l < k; l++)
        C[threadRowID * n + threadColID] += A[threadRowID * k + l] * B[l * n + threadColID];

    // int i, j, l;
    // for (i = 0; i < m; i++)
    // {
    //     for (l = 0; l < k; l++)
    //     {
    //         for (j = 0; j < n; j++)
    //         {
    //             C[i * n + j] += A[i * k + l] * B[l * n + j];
    //         }
    //     }
    // }
}

// The matrix sizes of A and B are m×k and k×n, respectively, so that C has size m×n
void matmult_gpu2(int m, int n, int k, double* A_h, double* B_h, double* C_h)
{
    // Allocate A_d, B_d, C_d
    double* A_d, * B_d, * C_d;
    hipMalloc((void**)&A_d, m * k * sizeof(double));
    hipMalloc((void**)&B_d, k * n * sizeof(double));
    hipMalloc((void**)&C_d, m * n * sizeof(double));

    // Transfer data
    hipMemcpy(A_d, A_h, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, k * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(C_d, 0, m * n * sizeof(int));

    // Launch kernel and synchronize
    int bs = BLOCK_SIZE; // TODO: if bs too large, doesn't work for small matrices
    int bsx = (m + (bs - 1)) / bs;
    int bsy = (n + (bs - 1)) / bs;
    dim3 dimGrid(bsx, bsy, 1);
    dim3 dimBlock(bs, bs, 1);
    gpu2_kernel << <dimGrid, dimBlock >> > (m, n, k, A_d, B_d, C_d, bsx, bsy);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(A_h, A_d, m * k * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(B_h, B_d, k * n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C_h, C_d, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Free A_d, B_d, C_d
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

}